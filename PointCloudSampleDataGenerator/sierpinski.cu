#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <fstream>
#include <iostream>

#define NUM_POINTS 10000000l
#define BLOCK_SIZE 512

struct Point {
    float x, y, z;
};

struct Color {
    float r, g, b;
};

__global__ void generatePoints(Point *points, Color *colors, long n) {
    unsigned long id = blockIdx.x * blockDim.x + threadIdx.x;
    hiprandState state;
    hiprand_init(id, id, 0, &state);

    Point vertices[4] = {{0.0, 0.0, 0.0}, {1.0, 0.0, 0.0}, {0.5, sqrtf(3.0)/2, 0.0}, {0.5, sqrtf(3.0)/6, sqrtf(2.0/3.0)}};
    Point p = {hiprand_uniform(&state), hiprand_uniform(&state), hiprand_uniform(&state)};

    if (id < n) {
        for (long i = 0; i < 10; ++i) { // Skip first few points
            int rnd = hiprand(&state) % 4;
            p.x = (p.x + vertices[rnd].x) / 2;
            p.y = (p.y + vertices[rnd].y) / 2;
            p.z = (p.z + vertices[rnd].z) / 2;
        }
        points[id] = p;

        // Assign color based on z-coordinate
        colors[id].r = p.z; // red component
        colors[id].g = 1.0 - p.z; // green component
        colors[id].b = p.x; // blue component
    }
}

int main() {
    Point *dev_points, *points;
    Color *dev_colors, *colors;
    
    points = (Point*)malloc(NUM_POINTS * sizeof(Point));
    colors = (Color*)malloc(NUM_POINTS * sizeof(Color));
    
    hipMalloc((void**)&dev_points, NUM_POINTS * sizeof(Point));
    hipMalloc((void**)&dev_colors, NUM_POINTS * sizeof(Color));

    dim3 blocksPerGrid((NUM_POINTS + BLOCK_SIZE - 1) / BLOCK_SIZE, 1, 1);
    dim3 threadsPerBlock(BLOCK_SIZE, 1, 1);

    generatePoints<<<blocksPerGrid, threadsPerBlock>>>(dev_points, dev_colors, NUM_POINTS);

    hipMemcpy(points, dev_points, NUM_POINTS * sizeof(Point), hipMemcpyDeviceToHost);
    hipMemcpy(colors, dev_colors, NUM_POINTS * sizeof(Color), hipMemcpyDeviceToHost);

    hipFree(dev_points);
    hipFree(dev_colors);

    std::ofstream output_file("sierpinski.csv");
    for (long i = 0; i < NUM_POINTS; i++)
        output_file << points[i].x << "," << points[i].y << "," << points[i].z << "," << colors[i].r << "," << colors[i].g << "," << colors[i].b << "\n";

    free(points);
    free(colors);

    return 0;
}

